#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define N   1024  // dimension size
#define M_PI   3.14159265358979323846  /* pi */
#define PRINT_FLAG 0
#define NPRINTS 30  // print size

void run_test_cufft_1d(int argc, char** argv) {
    // Declaration
    float *samples;
    hipfftComplex *complex_samples;
    hipfftComplex *complex_freq;
    hipfftComplex *d_complex_samples;
    hipfftComplex *d_complex_freq;
    hipfftHandle plan;

    size_t size = sizeof(hipfftComplex) * N;

    // Allocate memory for the variables on the host
    samples = (float *)malloc(sizeof(float) * N);
    complex_samples = (hipfftComplex *)malloc(size);
    complex_freq = (hipfftComplex *)malloc(size);

    // Input signal generation using cos(x)
    double delta = M_PI / 20.0;
    for (unsigned int i = 0; i < N; i++) {
        samples[i] = cos(i * delta);
    }

    // Convert to a complex signal
    for (unsigned int i = 0; i < N; i++) {
        complex_samples[i].x = samples[i];
        complex_samples[i].y = 0;
    }

    // Print input stuff
    if (PRINT_FLAG) {
        printf("Real data...\n");
        for (unsigned int i = 0; i < NPRINTS; i++) {
            printf("  %2.4f\n", samples[i]);
        }
        printf("Complex data...\n");
        for (unsigned int i = 0; i < NPRINTS; i++) {
            printf("  %2.4f + i%2.4f\n", complex_samples[i].x, complex_samples[i].y);
        }
    }

    // Allocate device memory for complex signal and output frequency
    CHECK_CUDA(hipMalloc((void **)&d_complex_samples, size));
    CHECK_CUDA(hipMalloc((void **)&d_complex_freq, size));

    // Copy host memory to device
    CHECK_CUDA(hipMemcpy(d_complex_samples, complex_samples, size, hipMemcpyHostToDevice));

    // Setup the CUFFT plan
    CHECK_CUFFT(hipfftPlan1d(&plan, N, HIPFFT_C2C, 1));
    
    // Execute a complex-to-complex 1D FFT
    CHECK_CUFFT(hipfftExecC2C(plan, d_complex_samples, d_complex_freq, HIPFFT_FORWARD));

    // Retrieve the results into host memory
    CHECK_CUDA(hipMemcpy(complex_freq, d_complex_freq, size, hipMemcpyDeviceToHost));

    // Print output stuff
    if (PRINT_FLAG) {
        printf("Fourier Coefficients...\n");
        for (unsigned int i = 0; i < NPRINTS; i++) {
            printf("  %2.4f + i%2.4f\n", complex_freq[i].x, complex_freq[i].y);
        }
    }

    // Clean up
    CHECK_CUFFT(hipfftDestroy(plan));
    CHECK_CUDA(hipFree(d_complex_freq));
    CHECK_CUDA(hipFree(d_complex_samples));
    free(complex_freq);
    free(complex_samples);
    free(samples);
}


int main(int argc, char **argv) {
    run_test_cufft_1d(argc, argv);
    CHECK_CUDA(hipDeviceReset());
    return 0;
}