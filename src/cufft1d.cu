#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define M_PI   3.14159265358979323846  /* pi */
#define PRINT_FLAG 0
#define NPRINTS 30  // print size

void run_test_cufft_1d(unsigned int nx) {
    // Declaration
    float *samples;
    hipfftComplex *complex_samples;
    hipfftComplex *complex_freq;
    hipfftComplex *d_complex_samples;
    hipfftComplex *d_complex_freq;
    hipfftHandle plan;

    size_t size = sizeof(hipfftComplex) * nx;

    hipEvent_t start, stop;
    float elapsed_time;

    // Allocate memory for the variables on the host
    samples = (float *)malloc(sizeof(float) * nx);
    complex_samples = (hipfftComplex *)malloc(size);
    complex_freq = (hipfftComplex *)malloc(size);

    // Input signal generation using cos(x)
    double delta = M_PI / 20.0;
    for (unsigned int i = 0; i < nx; i++) {
        samples[i] = cos(i * delta);
    }

    // Convert to a complex signal
    for (unsigned int i = 0; i < nx; i++) {
        complex_samples[i].x = samples[i];
        complex_samples[i].y = 0;
    }

    // Print input stuff
    if (PRINT_FLAG) {
        printf("Real data...\n");
        for (unsigned int i = 0; i < NPRINTS; i++) {
            printf("  %2.4f\n", samples[i]);
        }
        printf("Complex data...\n");
        for (unsigned int i = 0; i < NPRINTS; i++) {
            printf("  %2.4f + i%2.4f\n", complex_samples[i].x, complex_samples[i].y);
        }
    }

    // Create CUDA events
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Record the start event
    CHECK_CUDA(hipEventRecord(start, 0));

    // Allocate device memory for complex signal and output frequency
    CHECK_CUDA(hipMalloc((void **)&d_complex_samples, size));
    CHECK_CUDA(hipMalloc((void **)&d_complex_freq, size));

    // Copy host memory to device
    CHECK_CUDA(hipMemcpy(d_complex_samples, complex_samples, size, hipMemcpyHostToDevice));

    // Setup the CUFFT plan
    CHECK_CUFFT(hipfftPlan1d(&plan, nx, HIPFFT_C2C, 1));
    
    // Execute a complex-to-complex 1D FFT
    CHECK_CUFFT(hipfftExecC2C(plan, d_complex_samples, d_complex_freq, HIPFFT_FORWARD));

    // Retrieve the results into host memory
    CHECK_CUDA(hipMemcpy(complex_freq, d_complex_freq, size, hipMemcpyDeviceToHost));

    // Record the stop event
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));

    // Print output stuff
    if (PRINT_FLAG) {
        printf("Fourier Coefficients...\n");
        for (unsigned int i = 0; i < NPRINTS; i++) {
            printf("  %2.4f + i%2.4f\n", complex_freq[i].x, complex_freq[i].y);
        }
    }

    // Compute elapsed time
    CHECK_CUDA(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("%.6f\n", elapsed_time * 1e-3);

    // Clean up
    CHECK_CUFFT(hipfftDestroy(plan));
    CHECK_CUDA(hipFree(d_complex_freq));
    CHECK_CUDA(hipFree(d_complex_samples));
    free(complex_freq);
    free(complex_samples);
    free(samples);
}


int main(int argc, char **argv) {
    if (argc != 2) {
        printf("Error: This program requires exactly 1 command-line arguments.\n");
        return 1;
    }

    unsigned int nx = atoi(argv[1]);
    run_test_cufft_1d(nx);
    CHECK_CUDA(hipDeviceReset());
    return 0;
}