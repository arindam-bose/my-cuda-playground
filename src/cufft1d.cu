#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define N   1048576  // dimension size
#define M_PI   3.14159265358979323846  /* pi */
#define PRINT_FLAG 0
#define NPRINTS 30  // print size

void run_test_cufft_1d(int argc, char** argv) {
    // Declaration
    float *samples;
    hipfftComplex *complex_samples;
    hipfftComplex *complex_freq;
    hipfftComplex *d_complex_samples;
    hipfftComplex *d_complex_freq;
    hipfftHandle plan;

    // Allocate memory for the variables on the host
    samples = (float *)malloc(sizeof(float) * N);
    complex_samples = (hipfftComplex *)malloc(sizeof(hipfftComplex) * N);
    complex_freq = (hipfftComplex *)malloc(sizeof(hipfftComplex) * N);

    // Input signal generation using cos(x)
    double delta = M_PI / 20.0;
    for (unsigned int i = 0; i < N; i++) {
        samples[i] = cos(i * delta);
    }

    // Convert to a complex signal
    for (unsigned int i = 0; i < N; i++) {
        complex_samples[i].x = samples[i];
        complex_samples[i].y = 0;
    }

    // Print input stuff
    if (PRINT_FLAG) {
        printf("Real data...\n");
        for (unsigned int i = 0; i < NPRINTS; i++) {
            printf("  %2.4f\n", samples[i]);
        }
        printf("Complex data...\n");
        for (unsigned int i = 0; i < NPRINTS; i++) {
            printf("  %2.4f + i%2.4f\n", complex_samples[i].x, complex_samples[i].y);
        }
    }

    // Allocate device memory for complex signal and output frequency
    CHECK_CUDA(hipMalloc((void **)&d_complex_samples, sizeof(hipfftComplex) * N));
    CHECK_CUDA(hipMalloc((void **)&d_complex_freq, sizeof(hipfftComplex) * N));

    // Copy host memory to device
    CHECK_CUDA(hipMemcpy(d_complex_samples, complex_samples, sizeof(hipfftComplex) * N, hipMemcpyHostToDevice));

    // Setup the CUFFT plan
    CHECK_CUFFT(hipfftPlan1d(&plan, N, HIPFFT_C2C, 1));
    
    // Execute a complex-to-complex 1D FFT
    CHECK_CUFFT(hipfftExecC2C(plan, d_complex_samples, d_complex_freq, HIPFFT_FORWARD));

    // Retrieve the results into host memory
    CHECK_CUDA(hipMemcpy(complex_freq, d_complex_freq, sizeof(hipfftComplex) * N, hipMemcpyDeviceToHost));
    
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipDeviceReset());

    // Print output stuff
    if (PRINT_FLAG) {
        printf("Fourier Coefficients...\n");
        for (unsigned int i = 0; i < NPRINTS; i++) {
            printf("  %2.4f + i%2.4f\n", complex_freq[i].x, complex_freq[i].y);
        }
    }

    // Cleanups
    CHECK_CUFFT(hipfftDestroy(plan));
    CHECK_CUDA(hipFree(d_complex_freq));
    CHECK_CUDA(hipFree(d_complex_samples));
    free(complex_freq);
    free(complex_samples);
    free(samples);
}


int main(int argc, char **argv) {
    run_test_cufft_1d(argc, argv);
    return 0;
}