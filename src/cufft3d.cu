#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define NX 64  // X-dimension size
#define NY 64  // Y-dimension size
#define NZ 64  // Z-dimension size
#define ELEM_SIZE NX * NY * NZ

void run_test_cufft_3d(int argc, char** argv) {
    // Declaration
    hipfftComplex *complex_samples, *new_complex_samples;
    hipfftComplex *complex_freq;
    hipfftComplex *d_complex_samples;
    hipfftComplex *d_complex_freq;
    hipfftHandle plan;
    srand(2025);

    // Allocate memory for the variables on the host
    complex_samples = (hipfftComplex *)malloc(sizeof(hipfftComplex) * ELEM_SIZE);
    complex_freq = (hipfftComplex *)malloc(sizeof(hipfftComplex) * ELEM_SIZE);
    new_complex_samples = (hipfftComplex *)malloc(sizeof(hipfftComplex) * ELEM_SIZE);

    for (unsigned int i = 0; i < ELEM_SIZE; i++) {
        complex_samples[i].x = rand() / (float)RAND_MAX;
        complex_samples[i].y = 0;
    }

    // Allocate device memory for complex signal and output frequency
    hipMalloc((void **)&d_complex_samples, sizeof(hipfftComplex) * ELEM_SIZE);
    hipMalloc((void **)&d_complex_freq, sizeof(hipfftComplex) * ELEM_SIZE);

    // Copy host memory to device
    hipMemcpy(d_complex_samples, complex_samples, sizeof(hipfftComplex) * ELEM_SIZE, hipMemcpyHostToDevice);

    // Setup a 3D FFT plan
    hipfftPlan3d(&plan, NX, NY, NZ, HIPFFT_C2C);

    // Execute the forward 3D FFT (in-place computation)
    hipfftExecC2C(plan, d_complex_samples, d_complex_freq, HIPFFT_FORWARD);

    // Retrieve the results into host memory
    hipMemcpy(complex_freq, d_complex_freq, sizeof(hipfftComplex) * ELEM_SIZE, hipMemcpyDeviceToHost);

    // Execute the inverse 3D FFT (in-place computation)
    hipfftExecC2C(plan, d_complex_freq, d_complex_samples, HIPFFT_BACKWARD);

    // Retrieve the results into host memory
    hipMemcpy(new_complex_samples, d_complex_samples, sizeof(hipfftComplex) * ELEM_SIZE, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    for (unsigned int i = 0; i < ELEM_SIZE; i++) {
        new_complex_samples[i].x /= (float)ELEM_SIZE;
        new_complex_samples[i].y /= (float)ELEM_SIZE;
    }

    for (unsigned int i = 0; i < 30; i++) {
        printf("  %2.4f + i%2.4f -> %2.4f + i%2.4f\n", complex_samples[i].x, complex_samples[i].y, new_complex_samples[i].x, new_complex_samples[i].y);
    }

    // Clean up
    hipfftDestroy(plan);
    hipFree(d_complex_freq);
    hipFree(d_complex_samples);
    free(new_complex_samples);
    free(complex_freq);
    free(complex_samples);
}


int main(int argc, char **argv) {
    run_test_cufft_3d(argc, argv);
    return 0;
}