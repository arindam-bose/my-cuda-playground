#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define PRINT_FLAG 0
#define NPRINTS 30  // print size
#define IFFT_FLAG 0

void run_test_cufft_3d(unsigned int nx, unsigned int ny, unsigned int nz) {
    srand(2025);

    // Declaration
    hipfftComplex *complex_samples;
    hipfftComplex *new_complex_samples;
    hipfftComplex *complex_freq;
    hipfftComplex *d_complex_samples;
    hipfftComplex *d_complex_freq;
    hipfftHandle plan;

    unsigned int element_size = nx * ny * nz;

    // Allocate memory for the variables on the host
    complex_samples = (hipfftComplex *)malloc(sizeof(hipfftComplex) * element_size);
    complex_freq = (hipfftComplex *)malloc(sizeof(hipfftComplex) * element_size);
    if (IFFT_FLAG) {new_complex_samples = (hipfftComplex *)malloc(sizeof(hipfftComplex) * element_size);}

    // Initialize input complex signal
    for (unsigned int i = 0; i < element_size; i++) {
        complex_samples[i].x = rand() / (float)RAND_MAX;
        complex_samples[i].y = 0;
    }

    // Allocate device memory for complex signal and output frequency
    hipMalloc((void **)&d_complex_samples, sizeof(hipfftComplex) * element_size);
    hipMalloc((void **)&d_complex_freq, sizeof(hipfftComplex) * element_size);

    // Copy host memory to device
    hipMemcpy(d_complex_samples, complex_samples, sizeof(hipfftComplex) * element_size, hipMemcpyHostToDevice);

    // Setup a 3D FFT plan
    hipfftPlan3d(&plan, nx, ny, nz, HIPFFT_C2C);

    // Execute the forward 3D FFT (in-place computation)
    hipfftExecC2C(plan, d_complex_samples, d_complex_freq, HIPFFT_FORWARD);

    // Retrieve the results into host memory
    hipMemcpy(complex_freq, d_complex_freq, sizeof(hipfftComplex) * element_size, hipMemcpyDeviceToHost);

    if (IFFT_FLAG) {
        // Execute the inverse 3D IFFT (in-place computation)
        hipfftExecC2C(plan, d_complex_freq, d_complex_samples, HIPFFT_BACKWARD);

        // Retrieve the results into host memory
        hipMemcpy(new_complex_samples, d_complex_samples, sizeof(hipfftComplex) * element_size, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        // Normalize
        for (unsigned int i = 0; i < element_size; i++) {
            new_complex_samples[i].x /= (float)element_size;
            new_complex_samples[i].y /= (float)element_size;
        }
    }

    if (PRINT_FLAG && IFFT_FLAG) {
        printf("Complex samples after FFT and IFFT...\n");
        for (unsigned int i = 0; i < NPRINTS; i++) {
            printf("  %2.4f + i%2.4f -> %2.4f + i%2.4f\n", complex_samples[i].x, complex_samples[i].y, new_complex_samples[i].x, new_complex_samples[i].y);
        }
    }

    // Clean up
    hipfftDestroy(plan);
    hipFree(d_complex_freq);
    hipFree(d_complex_samples);
    if (IFFT_FLAG) {free(new_complex_samples);}
    free(complex_freq);
    free(complex_samples);
}


int main(int argc, char **argv) {
    if (argc != 4) {
        printf("Error: This program requires exactly 3 command-line arguments.\n");
        return 1;
    }

    int nx = atoi(argv[1]);
    int ny = atoi(argv[2]);
    int nz = atoi(argv[3]);
    run_test_cufft_3d(nx, ny, nz);
    return 0;
}