#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define NX 64  // X-dimension size
#define NY 64  // Y-dimension size
#define NZ 64  // Z-dimension size

int main() {
    hipfftHandle plan;
    hipfftComplex *d_data;
    size_t size = NX * NY * NZ * sizeof(hipfftComplex);

    // Allocate device memory for 3D data
    hipMalloc((void**)&d_data, size);

    // Create a 3D FFT plan
    if (hipfftPlan3d(&plan, NX, NY, NZ, HIPFFT_C2C) != HIPFFT_SUCCESS) {
        printf("CUFFT plan creation failed!\n");
        return -1;
    }

    // Execute the forward FFT (in-place computation)
    if (hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
        printf("CUFFT forward execution failed!\n");
        return -1;
    }

    // Execute the inverse FFT (to recover original data)
    if (hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
        printf("CUFFT inverse execution failed!\n");
        return -1;
    }

    // Normalize the output (since inverse FFT scales the result by NX*NY*NZ)
    int total_elements = NX * NY * NZ;
    hipDeviceSynchronize();

    printf("3D FFT execution completed successfully!\n");

    // Clean up
    hipfftDestroy(plan);
    hipFree(d_data);

    return 0;
}