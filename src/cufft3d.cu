#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define PRINT_FLAG 0
#define NPRINTS 30  // print size
#define IFFT_FLAG 0

void run_test_cufft_3d(unsigned int nx, unsigned int ny, unsigned int nz) {
    srand(2025);

    // Declaration
    hipfftComplex *complex_samples, *new_complex_samples;
    hipfftComplex *complex_freq;
    hipfftComplex *d_complex_samples;
    hipfftComplex *d_complex_freq;
    hipfftHandle plan;

    unsigned int element_size = nx * ny * nz;
    size_t size = sizeof(hipfftComplex) * element_size;

    // Allocate memory for the variables on the host
    complex_samples = (hipfftComplex *)malloc(size);
    complex_freq = (hipfftComplex *)malloc(size);
    if (IFFT_FLAG) {new_complex_samples = (hipfftComplex *)malloc(size);}

    // Initialize input complex signal
    for (unsigned int i = 0; i < element_size; i++) {
        complex_samples[i].x = rand() / (float)RAND_MAX;
        complex_samples[i].y = 0;
    }

    // Allocate device memory for complex signal and output frequency
    CHECK_CUDA(hipMalloc((void **)&d_complex_samples, size));
    CHECK_CUDA(hipMalloc((void **)&d_complex_freq, size));

    // Copy host memory to device
    CHECK_CUDA(hipMemcpy(d_complex_samples, complex_samples, size, hipMemcpyHostToDevice));

    // Setup a 3D FFT plan
    CHECK_CUFFT(hipfftPlan3d(&plan, nx, ny, nz, HIPFFT_C2C));

    // Execute the forward 3D FFT (in-place computation)
    CHECK_CUFFT(hipfftExecC2C(plan, d_complex_samples, d_complex_freq, HIPFFT_FORWARD));

    // Retrieve the results into host memory
    CHECK_CUDA(hipMemcpy(complex_freq, d_complex_freq, size, hipMemcpyDeviceToHost));

    if (IFFT_FLAG) {
        // Execute the inverse 3D IFFT (in-place computation)
        CHECK_CUFFT(hipfftExecC2C(plan, d_complex_freq, d_complex_samples, HIPFFT_BACKWARD));

        // Retrieve the results into host memory
        CHECK_CUDA(hipMemcpy(new_complex_samples, d_complex_samples, size, hipMemcpyDeviceToHost));

        // Normalize
        for (unsigned int i = 0; i < element_size; i++) {
            new_complex_samples[i].x /= (float)element_size;
            new_complex_samples[i].y /= (float)element_size;
        }
    }

    if (PRINT_FLAG && IFFT_FLAG) {
        printf("Complex samples after FFT and IFFT...\n");
        for (unsigned int i = 0; i < NPRINTS; i++) {
            printf("  %2.4f + i%2.4f -> %2.4f + i%2.4f\n", complex_samples[i].x, complex_samples[i].y, new_complex_samples[i].x, new_complex_samples[i].y);
        }
    }

    // Clean up
    CHECK_CUFFT(hipfftDestroy(plan));
    CHECK_CUDA(hipFree(d_complex_freq));
    CHECK_CUDA(hipFree(d_complex_samples));
    if (IFFT_FLAG) {free(new_complex_samples);}
    free(complex_freq);
    free(complex_samples);
}


int main(int argc, char **argv) {
    if (argc != 4) {
        printf("Error: This program requires exactly 3 command-line arguments.\n");
        return 1;
    }

    int nx = atoi(argv[1]);
    int ny = atoi(argv[2]);
    int nz = atoi(argv[3]);
    run_test_cufft_3d(nx, ny, nz);
    CHECK_CUDA(hipDeviceReset());
    return 0;
}