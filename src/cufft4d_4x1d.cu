#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define PRINT_FLAG 0
#define NPRINTS 30  // print size
#define NITER 5 // no. of iterations

// Function to execute 1D FFT along a specific dimension
void execute_cufft1d(hipfftComplex *d_idata, hipfftComplex *d_odata, int dim_size, int batch, int stride, int dist) {
    hipfftHandle plan;
    CHECK_CUFFT(hipfftPlanMany(&plan, 1, &dim_size, 
                                NULL, stride, dist, 
                                NULL, stride, dist, 
                                HIPFFT_C2C, batch));

    // Perform FFT
    CHECK_CUFFT(hipfftExecC2C(plan, d_idata, d_odata, HIPFFT_FORWARD));
    CHECK_CUFFT(hipfftDestroy(plan));
}

void run_test_cufft_4d(unsigned int nx, unsigned int ny, unsigned int nz, unsigned int nw) {
    srand(2025);

    // Declaration
    hipfftComplex *complex_samples;
    hipfftComplex *complex_freq;
    hipfftComplex *d_complex_samples;
    hipfftComplex *d_complex_freq;

    unsigned int element_size = nx * ny * nz * nw;
    size_t size = sizeof(hipfftComplex) * element_size;

    hipEvent_t start, stop;
    float elapsed_time;

    // Allocate memory for the variables on the host
    complex_samples = (hipfftComplex *)malloc(size);
    complex_freq = (hipfftComplex *)malloc(size);

    // Initialize input complex signal
    for (unsigned int i = 0; i < element_size; ++i) {
        complex_samples[i].x = rand() / (float)RAND_MAX;
        complex_samples[i].y = 0;
    }

    // Print input stuff
    if (PRINT_FLAG) {
        printf("Complex data...\n");
        for (unsigned int i = 0; i < NPRINTS; ++i) {
            printf("  %2.4f + i%2.4f\n", complex_samples[i].x, complex_samples[i].y);
        }
    }

    // Create CUDA events
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Record the start event
    CHECK_CUDA(hipEventRecord(start, 0));

    // Allocate device memory for complex signal and output frequency
    CHECK_CUDA(hipMalloc((void **)&d_complex_samples, size));
    CHECK_CUDA(hipMalloc((void **)&d_complex_freq, size));

    // Copy host memory to device
    CHECK_CUDA(hipMemcpy(d_complex_samples, complex_samples, size, hipMemcpyHostToDevice));

    // Perform FFT along each dimension sequentially
    execute_cufft1d(d_complex_samples, d_complex_freq, nx, ny * nz * nw, 1, nx);         // FFT along X
    execute_cufft1d(d_complex_freq, d_complex_freq, ny, nx * nz * nw, nx, ny);           // FFT along Y
    execute_cufft1d(d_complex_freq, d_complex_freq, nz, nx * ny * nw, nx * ny, nz);      // FFT along Z
    execute_cufft1d(d_complex_freq, d_complex_freq, nw, nx * ny * nz, nx * ny * nz, nw); // FFT along W

    // Retrieve the results into host memory
    CHECK_CUDA(hipMemcpy(complex_freq, d_complex_freq, size, hipMemcpyDeviceToHost));

    // Record the stop event
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));

    // Print output stuff
    if (PRINT_FLAG) {
        printf("Fourier Coefficients...\n");
        for (unsigned int i = 0; i < NPRINTS; ++i) {
            printf("  %2.4f + i%2.4f\n", complex_freq[i].x, complex_freq[i].y);
        }
    }

    // Compute elapsed time
    CHECK_CUDA(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("%.6f\n", elapsed_time * 1e-3);

    // Clean up
    CHECK_CUDA(hipFree(d_complex_freq));
    CHECK_CUDA(hipFree(d_complex_samples));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    free(complex_freq);
    free(complex_samples);
}


int main(int argc, char **argv) {
    if (argc != 5) {
        printf("Error: This program requires exactly 4 command-line arguments.\n");
        printf("       %s <arg0> <arg1> <arg2> <arg3>\n", argv[0]);
        printf("       arg0, arg1, arg2, arg3: FFT lengths in 4D\n");
        printf("       e.g.: %s 64 64 64 64\n", argv[0]);
        return -1;
    }

    unsigned int nx = atoi(argv[1]);
    unsigned int ny = atoi(argv[2]);
    unsigned int nz = atoi(argv[3]);
    unsigned int nw = atoi(argv[4]);
    run_test_cufft_4d(nx, ny, nz, nw);
    CHECK_CUDA(hipDeviceReset());
    return 0;
}