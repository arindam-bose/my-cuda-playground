#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define PRINT_FLAG 0
#define NPRINTS 30  // print size

// Function to execute 1D FFT along a specific dimension
void execute_cufft1d(hipfftComplex *d_idata, hipfftComplex *d_odata, int dim_size, int batch, int stride, int dist) {
    hipfftHandle plan;
    CHECK_CUFFT(hipfftPlanMany(&plan, 1, &dim_size, 
                                NULL, stride, dist, 
                                NULL, stride, dist, 
                                HIPFFT_C2C, batch));

    // Perform FFT
    CHECK_CUFFT(hipfftExecC2C(plan, d_idata, d_odata, HIPFFT_FORWARD));
    CHECK_CUFFT(hipfftDestroy(plan));
}

float run_test_cufft_4d_4x1d(unsigned int nx, unsigned int ny, unsigned int nz, unsigned int nw) {
    srand(2025);

    // Declaration
    hipfftComplex *complex_samples;
    hipfftComplex *complex_freq;
    hipfftComplex *d_complex_samples;
    hipfftComplex *d_complex_freq;

    unsigned int element_size = nx * ny * nz * nw;
    size_t size = sizeof(hipfftComplex) * element_size;

    hipEvent_t start, stop;
    float elapsed_time;

    // Allocate memory for the variables on the host
    complex_samples = (hipfftComplex *)malloc(size);
    complex_freq = (hipfftComplex *)malloc(size);

    // Initialize input complex signal
    for (unsigned int i = 0; i < element_size; ++i) {
        complex_samples[i].x = rand() / (float)RAND_MAX;
        complex_samples[i].y = 0;
    }

    // Print input stuff
    if (PRINT_FLAG) {
        printf("Complex data...\n");
        for (unsigned int i = 0; i < NPRINTS; ++i) {
            printf("  %2.4f + i%2.4f\n", complex_samples[i].x, complex_samples[i].y);
        }
    }

    // Create CUDA events
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Record the start event
    CHECK_CUDA(hipEventRecord(start, 0));

    // Allocate device memory for complex signal and output frequency
    CHECK_CUDA(hipMalloc((void **)&d_complex_samples, size));
    CHECK_CUDA(hipMalloc((void **)&d_complex_freq, size));

    // Copy host memory to device
    CHECK_CUDA(hipMemcpy(d_complex_samples, complex_samples, size, hipMemcpyHostToDevice));

    // Perform FFT along each dimension sequentially
    execute_cufft1d(d_complex_samples, d_complex_freq, nx, ny * nz * nw, 1, nx);         // FFT along X
    execute_cufft1d(d_complex_freq, d_complex_freq, ny, nx * nz * nw, nx, ny);           // FFT along Y
    execute_cufft1d(d_complex_freq, d_complex_freq, nz, nx * ny * nw, nx * ny, nz);      // FFT along Z
    execute_cufft1d(d_complex_freq, d_complex_freq, nw, nx * ny * nz, nx * ny * nz, nw); // FFT along W

    // Retrieve the results into host memory
    CHECK_CUDA(hipMemcpy(complex_freq, d_complex_freq, size, hipMemcpyDeviceToHost));

    // Record the stop event
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));

    // Print output stuff
    if (PRINT_FLAG) {
        printf("Fourier Coefficients...\n");
        for (unsigned int i = 0; i < NPRINTS; ++i) {
            printf("  %2.4f + i%2.4f\n", complex_freq[i].x, complex_freq[i].y);
        }
    }

    // Compute elapsed time
    CHECK_CUDA(hipEventElapsedTime(&elapsed_time, start, stop));
    // printf("%.6f\n", elapsed_time * 1e-3);

    // Clean up
    CHECK_CUDA(hipFree(d_complex_freq));
    CHECK_CUDA(hipFree(d_complex_samples));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    free(complex_freq);
    free(complex_samples);

    return elapsed_time * 1e-3;
}


int main(int argc, char **argv) {
    if (argc != 6) {
        printf("Error: This program requires exactly 5 command-line arguments.\n");
        printf("       %s <arg0> <arg1> <arg2> <arg3> <arg4>\n", argv[0]);
        printf("       arg0, arg1, arg2, arg3: FFT lengths in 4D\n");
        printf("       arg4: Number of iterations\n");
        printf("       e.g.: %s 64 64 64 64 5\n", argv[0]);
        return -1;
    }

    unsigned int nx = atoi(argv[1]);
    unsigned int ny = atoi(argv[2]);
    unsigned int nz = atoi(argv[3]);
    unsigned int nw = atoi(argv[4]);
    unsigned int niter = atoi(argv[5]);

    // Discard the first time running. It apparantly does some extra work during first time
    // JIT??
    run_test_cufft_4d_4x1d(nx, ny, nz, nw);

    float sum = 0.0;
    for (unsigned int i = 0; i < niter; ++i) {
        sum += run_test_cufft_4d_4x1d(nx, ny, nz, nw);
    }
    printf("%.6f\n", sum/(float)niter);

    CHECK_CUDA(hipDeviceReset());
    return 0;
}