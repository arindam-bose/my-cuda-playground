#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define PRINT_FLAG 1
#define NPRINTS 5  // print size

void printf_cufft_cmplx_array(hipfftComplex *complex_array, unsigned int size) {
    for (unsigned int i = 0; i < NPRINTS; ++i) {
        printf("  (%2.4f, %2.4fi)\n", complex_array[i].x, complex_array[i].y);
    }
    printf("...\n");
    for (unsigned int i = size - NPRINTS; i < size; ++i) {
        printf("  (%2.4f, %2.4fi)\n", complex_array[i].x, complex_array[i].y);
    }
}

void execute_cufft1d(hipfftComplex *d_idata, hipfftComplex *d_odata, int *dim, int *embed, int stride, int dist, int batch) {
    hipfftHandle plan;
    CHECK_CUFFT(hipfftPlanMany(&plan, 1, dim, 
                                embed, stride, dist, 
                                embed, stride, dist, 
                                HIPFFT_C2C, batch));

    // Perform FFT
    CHECK_CUFFT(hipfftExecC2C(plan, d_idata, d_odata, HIPFFT_FORWARD));
    CHECK_CUFFT(hipfftDestroy(plan));
}

float run_test_cufft_4d_4x1d(unsigned int nx, unsigned int ny, unsigned int nz, unsigned int nw) {
    srand(2025);

    // Declaration
    hipfftComplex *complex_data;
    hipfftComplex *d_complex_data;

    unsigned int element_size = nx * ny * nz * nw;
    size_t size = sizeof(hipfftComplex) * element_size;

    hipEvent_t start, stop;
    float elapsed_time;

    // Allocate memory for the variables on the host
    complex_data = (hipfftComplex *)malloc(size);

    // Initialize input complex signal
    for (unsigned int i = 0; i < element_size; ++i) {
        complex_data[i].x = rand() / (float)RAND_MAX;
        complex_data[i].y = 0;
    }

    // Print input stuff
    if (PRINT_FLAG) {
        printf("Complex data...\n");
        printf_cufft_cmplx_array(complex_data, element_size);
    }

    // Create CUDA events
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Allocate device memory for complex signal and output frequency
    CHECK_CUDA(hipMalloc((void **)&d_complex_data, size));

    hipfftHandle plan1d_x, plan1d_y, plan1d_z, plan1d_w;
    int n[1] = { (int)nx };
    CHECK_CUFFT(hipfftPlanMany(&plan1d_x, 1, n,       // 1D FFT of size nx
                            NULL, ny * nz * nw, nz, // inembed, istride, idist
                            NULL, ny * nz * nw, nx, // onembed, ostride, odist
                            HIPFFT_C2C, ny * nz * nw));
    n[0] = (int)ny;
    CHECK_CUFFT(hipfftPlanMany(&plan1d_y, 1, n,       // 1D FFT of size ny
                            NULL, nz * nw, ny, // inembed, istride, idist
                            NULL, nz * nw, ny, // onembed, ostride, odist
                            HIPFFT_C2C, nx * nz * nw));
    n[0] = (int)nz;
    CHECK_CUFFT(hipfftPlanMany(&plan1d_z, 1, n,       // 1D FFT of size nz
                            NULL, nw, nz, // inembed, istride, idist
                            NULL, nw, nz, // onembed, ostride, odist
                            HIPFFT_C2C, nx * ny * nw));
    n[0] = (int)nw;
    CHECK_CUFFT(hipfftPlanMany(&plan1d_w, 1, n,       // 1D FFT of size nw
                            NULL, 1, nw, // inembed, istride, idist
                            NULL, 1, nw, // onembed, ostride, odist
                            HIPFFT_C2C, nx * ny * nz));

    // Record the start event
    CHECK_CUDA(hipEventRecord(start, 0));

    // Copy host memory to device
    CHECK_CUDA(hipMemcpy(d_complex_data, complex_data, size, hipMemcpyHostToDevice));

    // Perform FFT along each dimension sequentially
    CHECK_CUFFT(hipfftExecC2C(plan1d_x, d_complex_data, d_complex_data, HIPFFT_FORWARD));
    CHECK_CUFFT(hipfftExecC2C(plan1d_y, d_complex_data, d_complex_data, HIPFFT_FORWARD));
    CHECK_CUFFT(hipfftExecC2C(plan1d_z, d_complex_data, d_complex_data, HIPFFT_FORWARD));
    // CHECK_CUFFT(hipfftExecC2C(plan1d_w, d_complex_data, d_complex_data, HIPFFT_FORWARD));

    // Retrieve the results into host memory
    CHECK_CUDA(hipMemcpy(complex_data, d_complex_data, size, hipMemcpyDeviceToHost));

    // Record the stop event
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));

    // Print output stuff
    if (PRINT_FLAG) {
        printf("Fourier Coefficients...\n");
        printf_cufft_cmplx_array(complex_data, element_size);
    }

    // Compute elapsed time
    CHECK_CUDA(hipEventElapsedTime(&elapsed_time, start, stop));

    // Clean up
    CHECK_CUFFT(hipfftDestroy(plan1d_w));
    CHECK_CUFFT(hipfftDestroy(plan1d_z));
    CHECK_CUFFT(hipfftDestroy(plan1d_y));
    CHECK_CUFFT(hipfftDestroy(plan1d_x));
    CHECK_CUDA(hipFree(d_complex_data));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    free(complex_data);

    return elapsed_time * 1e-3;
}


int main(int argc, char **argv) {
    if (argc != 6) {
        printf("Error: This program requires exactly 5 command-line arguments.\n");
        printf("       %s <arg0> <arg1> <arg2> <arg3> <arg4>\n", argv[0]);
        printf("       arg0, arg1, arg2, arg3: FFT lengths in 4D\n");
        printf("       arg4: Number of iterations\n");
        printf("       e.g.: %s 64 64 64 64 5\n", argv[0]);
        return -1;
    }

    unsigned int nx = atoi(argv[1]);
    unsigned int ny = atoi(argv[2]);
    unsigned int nz = atoi(argv[3]);
    unsigned int nw = atoi(argv[4]);
    unsigned int niter = atoi(argv[5]);

    // Discard the first time running. It apparantly does some extra work during first time
    // JIT??
    run_test_cufft_4d_4x1d(nx, ny, nz, nw);

    float sum = 0.0;
    for (unsigned int i = 0; i < niter; ++i) {
        sum += run_test_cufft_4d_4x1d(nx, ny, nz, nw);
    }
    printf("%.6f\n", sum/(float)niter);

    CHECK_CUDA(hipDeviceReset());
    return 0;
}