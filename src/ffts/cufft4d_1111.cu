#include "hip/hip_runtime.h"
#include "../../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h> 
#include <hipfft/hipfft.h>
#include <math.h>

#define PRINT_FLAG 1
#define NPRINTS 5  // print size

void printf_cufft_cmplx_array(hipfftComplex *complex_array, unsigned int size) {
    for (unsigned int i = 0; i < NPRINTS; ++i) {
        printf("  (%2.4f, %2.4fi)\n", complex_array[i].x, complex_array[i].y);
    }
    printf("...\n");
    for (unsigned int i = size - NPRINTS; i < size; ++i) {
        printf("  (%2.4f, %2.4fi)\n", complex_array[i].x, complex_array[i].y);
    }
}

// Function to execute 1D FFT along a specific dimension
void execute_fft(hipfftComplex *d_data, int dim_size, int batch_size) {
    hipfftHandle plan;
    int n[1] = { dim_size };
    int embed[1] = { dim_size };
    CHECK_CUFFT(hipfftPlanMany(&plan, 1, n, 
                            embed, 1, dim_size, 
                            embed, 1, dim_size, 
                            HIPFFT_C2C, batch_size));

    // Perform FFT
    CHECK_CUFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD));
    CHECK_CUFFT(hipfftDestroy(plan));
}


__global__ void do_circular_transpose(hipfftComplex *d_out, hipfftComplex *d_in, int nx, int ny, int nz, int nw) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int z = blockDim.z * blockIdx.z + threadIdx.z;

    if (x < nx && y < ny && z < nz) {
        for (int w = 0; w < nw; w++) {
            int in_idx  = ((x * ny + y) * nz + z) * nw + w;
            int out_idx = ((y * nz + z) * nw + w) * nx + x;
            d_out[out_idx] = d_in[in_idx];
        }
    }
}

float run_test_cufft_4d_4x1d(unsigned int nx, unsigned int ny, unsigned int nz, unsigned int nw) {
    srand(2025);

    // Declaration
    hipfftComplex *complex_data;
    hipfftComplex *d_complex_data;
    hipfftComplex *d_complex_data_swap;

    unsigned int element_size = nx * ny * nz * nw;
    size_t size = sizeof(hipfftComplex) * element_size;

    hipEvent_t start, stop;
    float elapsed_time;

    // Allocate memory for the variables on the host
    complex_data = (hipfftComplex *)malloc(size);

    // Initialize input complex signal
    for (unsigned int i = 0; i < element_size; ++i) {
        complex_data[i].x = rand() / (float)RAND_MAX;
        complex_data[i].y = 0;
    }

    // Print input stuff
    if (PRINT_FLAG) {
        printf("Complex data...\n");
        printf_cufft_cmplx_array(complex_data, element_size);
    }

    // Create CUDA events
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Allocate device memory for complex signal and output frequency
    CHECK_CUDA(hipMalloc((void **)&d_complex_data, size));
    CHECK_CUDA(hipMalloc((void **)&d_complex_data_swap, size));

    dim3 threads(8, 8, 8);
    dim3 blocks((nx + threads.x - 1) / threads.x, (ny + threads.y - 1) / threads.y, (nz + threads.z - 1) / threads.z);

    // Record the start event
    CHECK_CUDA(hipEventRecord(start, 0));

    // Copy host memory to device
    CHECK_CUDA(hipMemcpy(d_complex_data, complex_data, size, hipMemcpyHostToDevice));

    // Perform FFT along each dimension sequentially
    // Help from: https://forums.developer.nvidia.com/t/3d-and-4d-indexing-4d-fft/12564/2
    // and https://stackoverflow.com/questions/79574267/what-is-the-correct-way-to-perform-4d-fft-in-cuda-by-implementing-1d-fft-in-each

    // step 1: do 1-D FFT along w with number of element nw and batch=nx ny nz
    execute_fft(d_complex_data, nw, nx * ny * nz);
    // step 2: do tranpose operation A(x,y,z,w) → A(y,z,w,x)
    do_circular_transpose<<<blocks, threads>>>(d_complex_data_swap, d_complex_data, nx, ny, nz, nw);
    // step 3: do 1-D FFT along x with number of element nx and batch=n2n3n4
    execute_fft(d_complex_data_swap, nx, ny * nz * nw);
    // step 4: do tranpose operation A(y,z,w,x) → A(z,w,x,y)
    do_circular_transpose<<<blocks, threads>>>(d_complex_data, d_complex_data_swap, ny, nz, nw, nx);
    // step 5: do 1-D FFT along y with number of element ny and batch=n3n4n1
    execute_fft(d_complex_data, ny, nx * nz * nw);
    // step 6: do tranpose operation A(z,w,x,y) → A(w,x,y,z)
    do_circular_transpose<<<blocks, threads>>>(d_complex_data_swap, d_complex_data, nz, nw, nx, ny);
    // step 7: do 1-D FFT along z with number of element nz and batch=n4n1n2
    execute_fft(d_complex_data_swap, nz, nx * ny * nw);
    // step 8: do tranpose operation A(w,x,y,z) → A(x,y,z,w)
    do_circular_transpose<<<blocks, threads>>>(d_complex_data, d_complex_data_swap, nw, nx, ny, nz);

    // Retrieve the results into host memory
    CHECK_CUDA(hipMemcpy(complex_data, d_complex_data, size, hipMemcpyDeviceToHost));

    // Record the stop event
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));

    // Print output stuff
    if (PRINT_FLAG) {
        printf("Fourier Coefficients...\n");
        printf_cufft_cmplx_array(complex_data, element_size);
    }

    // Compute elapsed time
    CHECK_CUDA(hipEventElapsedTime(&elapsed_time, start, stop));

    // Clean up
    CHECK_CUDA(hipFree(d_complex_data));
    CHECK_CUDA(hipFree(d_complex_data_swap));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    free(complex_data);

    return elapsed_time * 1e-3;
}


int main(int argc, char **argv) {
    if (argc != 6) {
        printf("Error: This program requires exactly 5 command-line arguments.\n");
        printf("       %s <arg0> <arg1> <arg2> <arg3> <arg4>\n", argv[0]);
        printf("       arg0, arg1, arg2, arg3: FFT lengths in 4D\n");
        printf("       arg4: Number of iterations\n");
        printf("       e.g.: %s 64 64 64 64 5\n", argv[0]);
        return -1;
    }

    unsigned int nx = atoi(argv[1]);
    unsigned int ny = atoi(argv[2]);
    unsigned int nz = atoi(argv[3]);
    unsigned int nw = atoi(argv[4]);
    unsigned int niter = atoi(argv[5]);

    // Discard the first time running. It apparantly does some extra work during first time
    // JIT??
    run_test_cufft_4d_4x1d(nx, ny, nz, nw);

    float sum = 0.0;
    float span_s = 0.0;
    for (unsigned int i = 0; i < niter; ++i) {
        span_s = run_test_cufft_4d_4x1d(nx, ny, nz, nw);
        if (PRINT_FLAG) printf("[%d]: %.6f s\n", i, span_s);
        sum += span_s;
    }
    printf("%.6f\n", sum/(float)niter);

    CHECK_CUDA(hipDeviceReset());
    return 0;
}