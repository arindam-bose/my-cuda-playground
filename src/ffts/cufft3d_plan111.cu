#include "../../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define PRINT_FLAG 1
#define NPRINTS 16  // print size

void printf_cufft_cmplx_array(hipfftComplex *complex_array, unsigned int size) {
    for (unsigned int i = 0; i < NPRINTS; ++i) {
        printf("  (%2.4f, %2.4fi)\n", complex_array[i].x, complex_array[i].y);
    }
    printf("...\n");
    for (unsigned int i = size - NPRINTS; i < size; ++i) {
        printf("  (%2.4f, %2.4fi)\n", complex_array[i].x, complex_array[i].y);
    }
}

float run_test_cufft_3d(unsigned int nx, unsigned int ny, unsigned int nz) {
    srand(2025);

    // Declaration
    hipfftComplex *complex_data;
    hipfftComplex *d_complex_data;
    hipfftHandle plan1d_x, plan1d_y, plan1d_z;

    unsigned int element_size = nx * ny * nz;
    size_t size = sizeof(hipfftComplex) * element_size;

    hipEvent_t start, stop;
    float elapsed_time;

    // Allocate memory for the variables on the host
    complex_data = (hipfftComplex *)malloc(size);

    // Initialize input complex signal
    for (unsigned int i = 0; i < element_size; ++i) {
        complex_data[i].x = rand() / (float)RAND_MAX;
        complex_data[i].y = 0;
    }

    // Print input stuff
    if (PRINT_FLAG) {
        printf("Complex data...\n");
        printf_cufft_cmplx_array(complex_data, element_size);
    }
    
    // Create CUDA events
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Allocate device memory for complex signal and output frequency
    CHECK_CUDA(hipMalloc((void **)&d_complex_data, size));

    // Setup the CUFFT plans
    int n[1] = { (int)nx };
    int embed[1] = { (int)nx };
    CHECK_CUFFT(hipfftPlanMany(&plan1d_x, 1, n,          // 1D FFT of size nw
                            embed, ny * nz, 1,      // inembed, istride, idist
                            embed, ny * nz, 1,      // onembed, ostride, odist
                            HIPFFT_C2C, ny * 1));
    n[0] = (int)ny;
    embed[0] = (int)ny;
    CHECK_CUFFT(hipfftPlanMany(&plan1d_y, 1, n,          // 1D FFT of size nw
                            embed, nz, 1,      // inembed, istride, idist
                            embed, nz, 1,      // onembed, ostride, odist
                            HIPFFT_C2C, nx * nz));
    n[0] = (int)nz;
    embed[0] = (int)nz;
    CHECK_CUFFT(hipfftPlanMany(&plan1d_z, 1, n,          // 1D FFT of size nw
                            embed, 1, nz,      // inembed, istride, idist
                            embed, 1, nz,      // onembed, ostride, odist
                            HIPFFT_C2C, nx * ny));
    // Record the start event
    CHECK_CUDA(hipEventRecord(start, 0));

    // Copy host memory to device
    CHECK_CUDA(hipMemcpy(d_complex_data, complex_data, size, hipMemcpyHostToDevice));

    // Execute the forward 3D FFT (in-place computation)
    CHECK_CUFFT(hipfftExecC2C(plan1d_x, d_complex_data, d_complex_data, HIPFFT_FORWARD));
    CHECK_CUFFT(hipfftExecC2C(plan1d_y, d_complex_data, d_complex_data, HIPFFT_FORWARD));
    CHECK_CUFFT(hipfftExecC2C(plan1d_z, d_complex_data, d_complex_data, HIPFFT_FORWARD));

    // Retrieve the results into host memory
    CHECK_CUDA(hipMemcpy(complex_data, d_complex_data, size, hipMemcpyDeviceToHost));

    // Record the stop event
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));

    if (PRINT_FLAG) {
        printf("Fourier Coefficients...\n");
        printf_cufft_cmplx_array(complex_data, element_size);
    }

    // Compute elapsed time
    CHECK_CUDA(hipEventElapsedTime(&elapsed_time, start, stop));

    // Clean up
    CHECK_CUFFT(hipfftDestroy(plan1d_x));
    CHECK_CUFFT(hipfftDestroy(plan1d_y));
    CHECK_CUFFT(hipfftDestroy(plan1d_z));
    CHECK_CUDA(hipFree(d_complex_data));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    free(complex_data);

    return elapsed_time * 1e-3;
}


int main(int argc, char **argv) {
    if (argc != 5) {
        printf("Error: This program requires exactly 4 command-line arguments.\n");
        printf("       %s <arg0> <arg1> <arg2> <arg3>\n", argv[0]);
        printf("       arg0, arg1, arg2: FFT lengths in 3D\n");
        printf("       arg3: Number of iterations\n");
        printf("       e.g.: %s 64 64 64 5\n", argv[0]);
        return -1;
    }

    unsigned int nx = atoi(argv[1]);
    unsigned int ny = atoi(argv[2]);
    unsigned int nz = atoi(argv[3]);
    unsigned int niter = atoi(argv[4]);

    // Discard the first time running. It apparantly does some extra work during first time
    // JIT??
    run_test_cufft_3d(nx, ny, nz);

    float sum = 0.0;
    float span_s = 0.0;
    for (unsigned int i = 0; i < niter; ++i) {
        span_s = run_test_cufft_3d(nx, ny, nz);
        if (PRINT_FLAG) printf("[%d]: %.6f s\n", i, span_s);
        sum += span_s;
    }
    printf("%.6f\n", sum/(float)niter);

    CHECK_CUDA(hipDeviceReset());
    return 0;
}