#include "../../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <math.h>

#define PRINT_FLAG 1
#define NPRINTS 5  // print size

void printf_cufft_cmplx_array(hipfftComplex *complex_array, unsigned int size) {
    for (unsigned int i = 0; i < NPRINTS; ++i) {
        printf("  (%2.4f, %2.4fi)\n", complex_array[i].x, complex_array[i].y);
    }
    printf("...\n");
    for (unsigned int i = size - NPRINTS; i < size; ++i) {
        printf("  (%2.4f, %2.4fi)\n", complex_array[i].x, complex_array[i].y);
    }
}

float run_test_cufft_4d_3d1d(unsigned int nx, unsigned int ny, unsigned int nz, unsigned int nw) {
    srand(2025);
    
    // Declaration
    hipfftComplex *complex_data, *d_complex_data;
    hipfftHandle plan3d_xyz, plan1d_w;

    unsigned int element_size = nx * ny * nz * nw;
    size_t size = sizeof(hipfftComplex) * element_size;

    hipEvent_t start, stop;
    float elapsed_time;
    
    // Allocate memory for the variables on the host
    complex_data = (hipfftComplex *)malloc(size);

    // Initialize input complex signal
    for (unsigned int i = 0; i < element_size; ++i) {
        complex_data[i].x = rand() / (float)RAND_MAX;
        complex_data[i].y = 0;
    }

    // Print input stuff
    if (PRINT_FLAG) {
        printf("Complex data...\n");
        printf_cufft_cmplx_array(complex_data, element_size);
    }

    // Create CUDA events
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Allocate device memory for complex signal and output frequency
    CHECK_CUDA(hipMalloc((void **)&d_complex_data, size));

    // Setup FFT plans
    int n_xyz[3] = { (int)nx, (int)ny, (int)nz };
    int embed[3] = { (int)nx, (int)ny, (int)nz };
    CHECK_CUFFT(hipfftPlanMany(&plan3d_xyz, 3, n_xyz,          // 1D FFT of size nw
                            embed, nw, 1,     // inembed, istride, idist
                            embed, nw, 1,     // onembed, ostride, odist
                            HIPFFT_C2C, nw));
    int n_w[1] = { (int)nw };
    CHECK_CUFFT(hipfftPlanMany(&plan1d_w, 1, n_w,       // 1D FFT of size nw
                            NULL, 1, nw, // inembed, istride, idist
                            NULL, 1, nw, // onembed, ostride, odist
                            HIPFFT_C2C, nx * ny * nz));

    // Record the start event
    CHECK_CUDA(hipEventRecord(start, 0));

    // Copy host memory to device
    CHECK_CUDA(hipMemcpy(d_complex_data, complex_data, size, hipMemcpyHostToDevice));

    CHECK_CUFFT(hipfftExecC2C(plan3d_xyz, d_complex_data, d_complex_data, HIPFFT_FORWARD));
    CHECK_CUFFT(hipfftDestroy(plan3d_xyz));
    CHECK_CUFFT(hipfftExecC2C(plan1d_w, d_complex_data, d_complex_data, HIPFFT_FORWARD));
    CHECK_CUFFT(hipfftDestroy(plan1d_w));

    // Copy results back to host
    CHECK_CUDA(hipMemcpy(complex_data, d_complex_data, size, hipMemcpyDeviceToHost));

    // Record the stop event
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));

    // Print output stuff
    if (PRINT_FLAG) {
        printf("Fourier Coefficients...\n");
        printf_cufft_cmplx_array(complex_data, element_size);
    }

    // Compute elapsed time
    CHECK_CUDA(hipEventElapsedTime(&elapsed_time, start, stop));

    // Cleanup
    CHECK_CUDA(hipFree(d_complex_data));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    free(complex_data);

    return elapsed_time * 1e-3;
}


int main(int argc, char **argv) {
    if (argc != 6) {
        printf("Error: This program requires exactly 5 command-line arguments.\n");
        printf("       %s <arg0> <arg1> <arg2> <arg3> <arg4>\n", argv[0]);
        printf("       arg0, arg1, arg2, arg3: FFT lengths in 4D\n");
        printf("       arg4: Number of iterations\n");
        printf("       e.g.: %s 64 64 64 64 5\n", argv[0]);
        return -1;
    }

    unsigned int nx = atoi(argv[1]);
    unsigned int ny = atoi(argv[2]);
    unsigned int nz = atoi(argv[3]);
    unsigned int nw = atoi(argv[4]);
    unsigned int niter = atoi(argv[5]);

    // Discard the first time running. It apparantly does some extra work during first time
    // JIT??
    run_test_cufft_4d_3d1d(nx, ny, nz, nw);

    float sum = 0.0;
    float span_s = 0.0;
    for (unsigned int i = 0; i < niter; ++i) {
        span_s = run_test_cufft_4d_3d1d(nx, ny, nz, nw);
        if (PRINT_FLAG) printf("[%d]: %.6f s\n", i, span_s);
        sum += span_s;
    }
    printf("%.6f\n", sum/(float)niter);

    CHECK_CUDA(hipDeviceReset());
    return 0;
}