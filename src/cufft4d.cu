#include "../common/common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define NX 64  // X Dimension
#define NY 64  // Y Dimension
#define NZ 512  // Z Dimension
#define NW 128  // W Dimension
#define TOTAL_ELEMENTS (NX * NY * NZ * NW)

// Function to execute 1D FFT along a specific dimension
void execute_fft(hipfftComplex *d_data, int dim_size, int batch, int stride, int dist) {
    hipfftHandle plan;
    CHECK_CUFFT(hipfftPlanMany(&plan, 1, &dim_size, 
                              NULL, stride, dist, 
                              NULL, stride, dist, 
                              HIPFFT_C2C, batch));

    // Perform FFT
    CHECK_CUFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD));
    CHECK_CUFFT(hipfftDestroy(plan));
}

int main() {
    hipfftComplex *data, *d_data;
    size_t size = TOTAL_ELEMENTS * sizeof(hipfftComplex);

    data = (hipfftComplex *)malloc(size);
    for (unsigned int i = 0; i < TOTAL_ELEMENTS; i++) {
        data[i].x = rand() / (float)RAND_MAX;
        data[i].y = 0;
    }

    // Allocate device memory for 4D data
    CHECK_CUDA(hipMalloc((void**)&d_data, size));

    CHECK_CUDA(hipMemcpy(d_data, data, size, hipMemcpyHostToDevice));

    // Perform FFT along each dimension sequentially
    execute_fft(d_data, NX, NY * NZ * NW, 1, NX);            // FFT along X
    execute_fft(d_data, NY, NX * NZ * NW, NX, NY);           // FFT along Y
    execute_fft(d_data, NZ, NX * NY * NW, NX * NY, NZ);      // FFT along Z
    execute_fft(d_data, NW, NX * NY * NZ, NX * NY * NZ, NW); // FFT along W

    // Free GPU memory
    CHECK_CUDA(hipFree(d_data));

    printf("4D FFT execution completed successfully!\n");
    return 0;
}
