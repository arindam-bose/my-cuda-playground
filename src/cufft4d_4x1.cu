#include "../common/common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define PRINT_FLAG 0
#define NPRINTS 30  // print size

// Function to execute 1D FFT along a specific dimension
void execute_cufft1d(hipfftComplex *d_idata, hipfftComplex *d_odata, int dim_size, int batch, int stride, int dist) {
    hipfftHandle plan;
    CHECK_CUFFT(hipfftPlanMany(&plan, 1, &dim_size, 
                                NULL, stride, dist, 
                                NULL, stride, dist, 
                                HIPFFT_C2C, batch));

    // Perform FFT
    CHECK_CUFFT(hipfftExecC2C(plan, d_idata, d_odata, HIPFFT_FORWARD));
    CHECK_CUFFT(hipfftDestroy(plan));
}

void run_test_cufft_4d(unsigned int nx, unsigned int ny, unsigned int nz, unsigned int nw) {
    srand(2025);

    // Declaration
    hipfftComplex *complex_samples;
    hipfftComplex *complex_freq;
    hipfftComplex *d_complex_samples;
    hipfftComplex *d_complex_freq;

    unsigned int element_size = nx * ny * nz * nw;
    size_t size = sizeof(hipfftComplex) * element_size;

    // Allocate memory for the variables on the host
    complex_samples = (hipfftComplex *)malloc(size);
    complex_freq = (hipfftComplex *)malloc(size);

    // Initialize input complex signal
    for (unsigned int i = 0; i < element_size; i++) {
        complex_samples[i].x = rand() / (float)RAND_MAX;
        complex_samples[i].y = 0;
    }

    // Allocate device memory for complex signal and output frequency
    CHECK_CUDA(hipMalloc((void **)&d_complex_samples, size));
    CHECK_CUDA(hipMalloc((void **)&d_complex_freq, size));

    // Copy host memory to device
    CHECK_CUDA(hipMemcpy(d_complex_samples, complex_samples, size, hipMemcpyHostToDevice));

    // Perform FFT along each dimension sequentially
    execute_cufft1d(d_complex_samples, d_complex_freq, nx, ny * nz * nw, 1, nx);         // FFT along X
    execute_cufft1d(d_complex_freq, d_complex_freq, ny, nx * nz * nw, nx, ny);           // FFT along Y
    execute_cufft1d(d_complex_freq, d_complex_freq, nz, nx * ny * nw, nx * ny, nz);      // FFT along Z
    execute_cufft1d(d_complex_freq, d_complex_freq, nw, nx * ny * nz, nx * ny * nz, nw); // FFT along W

    // Retrieve the results into host memory
    CHECK_CUDA(hipMemcpy(complex_freq, d_complex_freq, size, hipMemcpyDeviceToHost));

    // Print output stuff
    if (PRINT_FLAG) {
        printf("Fourier Coefficients...\n");
        for (unsigned int i = 0; i < NPRINTS; i++) {
            printf("  %2.4f + i%2.4f\n", complex_freq[i].x, complex_freq[i].y);
        }
    }

    // Clean up
    CHECK_CUDA(hipFree(d_complex_freq));
    CHECK_CUDA(hipFree(d_complex_samples));
    free(complex_freq);
    free(complex_samples);
}

int main(int argc, char **argv) {
    if (argc != 5) {
        printf("Error: This program requires exactly 5 command-line arguments.\n");
        return 1;
    }

    int nx = atoi(argv[1]);
    int ny = atoi(argv[2]);
    int nz = atoi(argv[3]);
    int nw = atoi(argv[4]);
    run_test_cufft_4d(nx, ny, nz, nw);
    CHECK_CUDA(hipDeviceReset());
    return 0;
}