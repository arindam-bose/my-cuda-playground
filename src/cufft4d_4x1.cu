#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define PRINT_FLAG 0
#define NPRINTS 30  // print size

// Function to execute 1D FFT along a specific dimension
void execute_cufft1d(hipfftComplex *d_idata, hipfftComplex *d_odata, int dim_size, int batch, int stride, int dist) {
    hipfftHandle plan;
    CHECK_CUFFT(hipfftPlanMany(&plan, 1, &dim_size, 
                                NULL, stride, dist, 
                                NULL, stride, dist, 
                                HIPFFT_C2C, batch));

    // Perform FFT
    CHECK_CUFFT(hipfftExecC2C(plan, d_idata, d_odata, HIPFFT_FORWARD));
    CHECK_CUFFT(hipfftDestroy(plan));
}

void run_test_cufft_4d(unsigned int nx, unsigned int ny, unsigned int nz, unsigned int nw) {
    srand(2025);

    // Declaration
    hipfftComplex *complex_samples;
    hipfftComplex *complex_freq;
    hipfftComplex *d_complex_samples;
    hipfftComplex *d_complex_freq;

    unsigned int element_size = nx * ny * nz * nw;
    size_t size = sizeof(hipfftComplex) * element_size;

    hipEvent_t start, stop;
    float elapsed_time;

    // Allocate memory for the variables on the host
    complex_samples = (hipfftComplex *)malloc(size);
    complex_freq = (hipfftComplex *)malloc(size);

    // Initialize input complex signal
    for (unsigned int i = 0; i < element_size; i++) {
        complex_samples[i].x = rand() / (float)RAND_MAX;
        complex_samples[i].y = 0;
    }

    // Create CUDA events
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Record the start event
    CHECK_CUDA(hipEventRecord(start, 0));

    // Allocate device memory for complex signal and output frequency
    CHECK_CUDA(hipMalloc((void **)&d_complex_samples, size));
    CHECK_CUDA(hipMalloc((void **)&d_complex_freq, size));

    // Copy host memory to device
    CHECK_CUDA(hipMemcpy(d_complex_samples, complex_samples, size, hipMemcpyHostToDevice));

    // Perform FFT along each dimension sequentially
    execute_cufft1d(d_complex_samples, d_complex_freq, nx, ny * nz * nw, 1, nx);         // FFT along X
    execute_cufft1d(d_complex_freq, d_complex_freq, ny, nx * nz * nw, nx, ny);           // FFT along Y
    execute_cufft1d(d_complex_freq, d_complex_freq, nz, nx * ny * nw, nx * ny, nz);      // FFT along Z
    execute_cufft1d(d_complex_freq, d_complex_freq, nw, nx * ny * nz, nx * ny * nz, nw); // FFT along W

    // Retrieve the results into host memory
    CHECK_CUDA(hipMemcpy(complex_freq, d_complex_freq, size, hipMemcpyDeviceToHost));

    // Record the stop event
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));

    // Print output stuff
    if (PRINT_FLAG) {
        printf("Fourier Coefficients...\n");
        for (unsigned int i = 0; i < NPRINTS; i++) {
            printf("  %2.4f + i%2.4f\n", complex_freq[i].x, complex_freq[i].y);
        }
    }

    // Compute elapsed time
    CHECK_CUDA(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("Elapsed time: %.6f ms\n", elapsed_time);

    // Clean up
    CHECK_CUDA(hipFree(d_complex_freq));
    CHECK_CUDA(hipFree(d_complex_samples));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    free(complex_freq);
    free(complex_samples);
}


int main(int argc, char **argv) {
    if (argc != 5) {
        printf("Error: This program requires exactly 5 command-line arguments.\n");
        return 1;
    }

    int nx = atoi(argv[1]);
    int ny = atoi(argv[2]);
    int nz = atoi(argv[3]);
    int nw = atoi(argv[4]);
    run_test_cufft_4d(nx, ny, nz, nw);
    CHECK_CUDA(hipDeviceReset());
    return 0;
}