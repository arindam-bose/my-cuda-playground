#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <math.h>

#define PRINT_FLAG 1
#define NPRINTS 16  // print size

void printf_cufft_cmplx_array(hipfftComplex *complex_array, unsigned int size) {
    for (unsigned int i = 0; i < NPRINTS; ++i) {
        printf("  %2.4f + i%2.4f\n", complex_array[i].x, complex_array[i].y);
    }
    printf("...\n");
    for (unsigned int i = size - NPRINTS; i < size; ++i) {
        printf("  %2.4f + i%2.4f\n", complex_array[i].x, complex_array[i].y);
    }
}

float run_test_cufft_4d_3d1d(unsigned int nx, unsigned int ny, unsigned int nz, unsigned int nw) {
    srand(2025);
    
    // Declaration
    hipfftComplex *complex_data;
    hipfftComplex *d_complex_data;
    hipfftHandle plan3d, plan1d;

    unsigned int element_size = nx * ny * nz * nw;
    size_t size = sizeof(hipfftComplex) * element_size;

    hipEvent_t start, stop;
    float elapsed_time;
    
    // Allocate memory for the variables on the host
    complex_data = (hipfftComplex *)malloc(size);

    // Initialize input complex signal
    for (unsigned int i = 0; i < element_size; ++i) {
        complex_data[i].x = rand() / (float)RAND_MAX;
        complex_data[i].y = 0;
    }

    // Print input stuff
    if (PRINT_FLAG) {
        printf("Complex data...\n");
        printf_cufft_cmplx_array(complex_data, element_size);
    }

    // Create CUDA events
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Record the start event
    CHECK_CUDA(hipEventRecord(start, 0));

    // Allocate device memory for complex signal and output frequency
    CHECK_CUDA(hipMalloc((void **)&d_complex_data, size));

    // Copy host memory to device
    CHECK_CUDA(hipMemcpy(d_complex_data, complex_data, size, hipMemcpyHostToDevice));

    // -----------------------
    // 1. Perform 3D FFTs over each W slice (W batches of 3D volumes)
    // -----------------------
    CHECK_CUFFT(hipfftPlan3d(&plan3d, nx, ny, nz, HIPFFT_C2C));
    for (int w = 0; w < nw; ++w) {
        size_t offset = w * nx * ny * nz;
        CHECK_CUFFT(hipfftExecC2C(plan3d, d_complex_data + offset, d_complex_data + offset, HIPFFT_FORWARD));
    }

    // -----------------------
    // 2. Perform 1D FFT along W dimension
    // -----------------------
    // There are NX*NY*NZ such transforms (one for each (x,y,z) point)
    int n[1] = { (int)nw };
    int batch = nx * ny * nz;
    int embed[1] = { (int)nw };
    int stride = 1;
    int dist = nw;

    CHECK_CUFFT(hipfftPlanMany(&plan1d, 1, n,       // rank, dimensions
                                embed, stride, dist,
                                embed, stride, dist,
                                HIPFFT_C2C, batch));

    // Execute the 1D FFTs (in-place)
    CHECK_CUFFT(hipfftExecC2C(plan1d, d_complex_data, d_complex_data, HIPFFT_FORWARD));

    // Retrieve the results into host memory
    CHECK_CUDA(hipMemcpy(complex_data, d_complex_data, size, hipMemcpyDeviceToHost));

    // Record the stop event
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));

    // Print output stuff
    if (PRINT_FLAG) {
        printf("Fourier Coefficients...\n");
        printf_cufft_cmplx_array(complex_data, element_size);
    }

    // Compute elapsed time
    CHECK_CUDA(hipEventElapsedTime(&elapsed_time, start, stop));
    // printf("%.6f\n", elapsed_time * 1e-3);

    // Cleanup
    CHECK_CUDA(hipFree(d_complex_data));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    CHECK_CUFFT(hipfftDestroy(plan3d));
    CHECK_CUFFT(hipfftDestroy(plan1d));
    free(complex_data);

    return elapsed_time * 1e-3;
}


int main(int argc, char **argv) {
    if (argc != 6) {
        printf("Error: This program requires exactly 5 command-line arguments.\n");
        printf("       %s <arg0> <arg1> <arg2> <arg3> <arg4>\n", argv[0]);
        printf("       arg0, arg1, arg2, arg3: FFT lengths in 4D\n");
        printf("       arg4: Number of iterations\n");
        printf("       e.g.: %s 64 64 64 64 5\n", argv[0]);
        return -1;
    }

    unsigned int nx = atoi(argv[1]);
    unsigned int ny = atoi(argv[2]);
    unsigned int nz = atoi(argv[3]);
    unsigned int nw = atoi(argv[4]);
    unsigned int niter = atoi(argv[5]);

    // Discard the first time running. It apparantly does some extra work during first time
    // JIT??
    run_test_cufft_4d_3d1d(nx, ny, nz, nw);

    float sum = 0.0;
    for (unsigned int i = 0; i < niter; ++i) {
        sum += run_test_cufft_4d_3d1d(nx, ny, nz, nw);
    }
    printf("%.6f\n", sum/(float)niter);

    CHECK_CUDA(hipDeviceReset());
    return 0;
}