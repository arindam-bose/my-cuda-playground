#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <math.h>

#define PRINT_FLAG 0
#define NPRINTS 30  // print size

float run_test_cufft_4d_3d1d(unsigned int nx, unsigned int ny, unsigned int nz, unsigned int nw) {
    srand(2025);
    
    // Declaration
    hipfftComplex *complex_samples;
    hipfftComplex *complex_freq;
    hipfftComplex *d_complex_samples;
    hipfftComplex *d_complex_freq;
    hipfftHandle plan3d, plan1d;

    unsigned int element_size = nx * ny * nz * nw;
    size_t size = sizeof(hipfftComplex) * element_size;

    hipEvent_t start, stop;
    float elapsed_time;
    
    // Allocate memory for the variables on the host
    complex_samples = (hipfftComplex *)malloc(size);
    complex_freq = (hipfftComplex *)malloc(size);

    // Initialize input complex signal
    for (unsigned int i = 0; i < element_size; ++i) {
        complex_samples[i].x = rand() / (float)RAND_MAX;
        complex_samples[i].y = 0;
    }

    // Print input stuff
    if (PRINT_FLAG) {
        printf("Complex data...\n");
        for (unsigned int i = 0; i < NPRINTS; ++i) {
            printf("  %2.4f + i%2.4f\n", complex_samples[i].x, complex_samples[i].y);
        }
    }

    // Create CUDA events
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Record the start event
    CHECK_CUDA(hipEventRecord(start, 0));

    // Allocate device memory for complex signal and output frequency
    CHECK_CUDA(hipMalloc((void **)&d_complex_samples, size));
    CHECK_CUDA(hipMalloc((void **)&d_complex_freq, size));

    // Copy host memory to device
    CHECK_CUDA(hipMemcpy(d_complex_samples, complex_samples, size, hipMemcpyHostToDevice));

    // -----------------------
    // 1. Perform 3D FFTs over each W slice (W batches of 3D volumes)
    // -----------------------
    CHECK_CUFFT(hipfftPlan3d(&plan3d, nx, ny, nz, HIPFFT_C2C));
    for (int w = 0; w < nw; ++w) {
        size_t offset = w * nx * ny * nz;
        CHECK_CUFFT(hipfftExecC2C(plan3d, d_complex_samples + offset, d_complex_freq + offset, HIPFFT_FORWARD));
    }

    // -----------------------
    // 2. Perform 1D FFT along W dimension
    // -----------------------
    // There are NX*NY*NZ such transforms (one for each (x,y,z) point)
    int n[1] = { (int)nw };
    int batch = nx * ny * nz;
    int stride = 1;
    int dist = nw;

    CHECK_CUFFT(hipfftPlanMany(&plan1d, 1, n,       // rank, dimensions
                                NULL, stride, dist,
                                NULL, stride, dist,
                                HIPFFT_C2C, batch));

    // Execute the 1D FFTs (in-place)
    CHECK_CUFFT(hipfftExecC2C(plan1d, d_complex_freq, d_complex_freq, HIPFFT_FORWARD));

    // Retrieve the results into host memory
    CHECK_CUDA(hipMemcpy(complex_freq, d_complex_freq, size, hipMemcpyDeviceToHost));

    // Record the stop event
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));

    // Print output stuff
    if (PRINT_FLAG) {
        printf("Fourier Coefficients...\n");
        for (unsigned int i = 0; i < NPRINTS; ++i) {
            printf("  %2.4f + i%2.4f\n", complex_freq[i].x, complex_freq[i].y);
        }
    }

    // Compute elapsed time
    CHECK_CUDA(hipEventElapsedTime(&elapsed_time, start, stop));
    // printf("%.6f\n", elapsed_time * 1e-3);

    // Cleanup
    CHECK_CUDA(hipFree(d_complex_freq));
    CHECK_CUDA(hipFree(d_complex_samples));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    CHECK_CUFFT(hipfftDestroy(plan3d));
    CHECK_CUFFT(hipfftDestroy(plan1d));
    free(complex_freq);
    free(complex_samples);

    return elapsed_time * 1e-3;
}


int main(int argc, char **argv) {
    if (argc != 6) {
        printf("Error: This program requires exactly 5 command-line arguments.\n");
        printf("       %s <arg0> <arg1> <arg2> <arg3> <arg4>\n", argv[0]);
        printf("       arg0, arg1, arg2, arg3: FFT lengths in 4D\n");
        printf("       arg4: Number of iterations\n");
        printf("       e.g.: %s 64 64 64 64 5\n", argv[0]);
        return -1;
    }

    unsigned int nx = atoi(argv[1]);
    unsigned int ny = atoi(argv[2]);
    unsigned int nz = atoi(argv[3]);
    unsigned int nw = atoi(argv[4]);
    unsigned int niter = atoi(argv[5]);

    // Discard the first time running. It apparantly does some extra work during first time
    // JIT??
    run_test_cufft_4d_3d1d(nx, ny, nz, nw);

    float sum = 0.0;
    for (unsigned int i = 0; i < niter; ++i) {
        sum += run_test_cufft_4d_3d1d(nx, ny, nz, nw);
    }
    printf("%.6f\n", sum/(float)niter);

    CHECK_CUDA(hipDeviceReset());
    return 0;
}