#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define PRINT_FLAG 1
#define NPRINTS 5  // print size

void printf_cufft_cmplx_array(hipfftComplex *complex_array, unsigned int size) {
    for (unsigned int i = 0; i < NPRINTS; ++i) {
        printf("  %2.4f + i%2.4f\n", complex_array[i].x, complex_array[i].y);
    }
    printf("...\n");
    for (unsigned int i = size - NPRINTS; i < size; ++i) {
        printf("  %2.4f + i%2.4f\n", complex_array[i].x, complex_array[i].y);
    }
}

float run_test_cufft_2d(unsigned int nx, unsigned int ny) {
    srand(2025);

    // Declaration
    hipfftComplex *complex_data;
    hipfftComplex *d_complex_data;
    hipfftHandle plan;

    unsigned int element_size = nx * ny;
    size_t size = sizeof(hipfftComplex) * element_size;

    hipEvent_t start, stop;
    float elapsed_time;

    // Allocate memory for the variables on the host
    complex_data = (hipfftComplex *)malloc(size);

    // Initialize input complex signal
    for (unsigned int i = 0; i < element_size; ++i) {
        complex_data[i].x = rand() / (float)RAND_MAX;
        complex_data[i].y = 0;
    }

    // Print input stuff
    if (PRINT_FLAG) {
        printf("Complex data...\n");
        printf_cufft_cmplx_array(complex_data, element_size);
    }

    // Create CUDA events
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Record the start event
    CHECK_CUDA(hipEventRecord(start, 0));

    // Allocate device memory for complex signal and output frequency
    CHECK_CUDA(hipMalloc((void **)&d_complex_data, size));

    // Copy host memory to device
    CHECK_CUDA(hipMemcpy(d_complex_data, complex_data, size, hipMemcpyHostToDevice));

    // Setup the CUFFT plan
    CHECK_CUFFT(hipfftPlan2d(&plan, nx, ny, HIPFFT_C2C));
    
    // Execute a complex-to-complex 1D FFT
    CHECK_CUFFT(hipfftExecC2C(plan, d_complex_data, d_complex_data, HIPFFT_FORWARD));

    // Retrieve the results into host memory
    CHECK_CUDA(hipMemcpy(complex_data, d_complex_data, size, hipMemcpyDeviceToHost));

    // Record the stop event
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));

    // Print output stuff
    if (PRINT_FLAG) {
        printf("Fourier Coefficients...\n");
        printf_cufft_cmplx_array(complex_data, element_size);
    }

    // Compute elapsed time
    CHECK_CUDA(hipEventElapsedTime(&elapsed_time, start, stop));

    // Clean up
    CHECK_CUFFT(hipfftDestroy(plan));
    CHECK_CUDA(hipFree(d_complex_data));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    free(complex_data);

    return elapsed_time * 1e-3;
}


int main(int argc, char **argv) {
    if (argc != 4) {
        printf("Error: This program requires exactly 3 command-line arguments.\n");
        printf("       %s <arg0> <arg1> <arg2>\n", argv[0]);
        printf("       arg0, arg1: FFT lengths in 2D\n");
        printf("       arg2: Number of iterations\n");
        printf("       e.g.: %s 64 64 5\n", argv[0]);
        return -1;
    }

    unsigned int nx = atoi(argv[1]);
    unsigned int ny = atoi(argv[2]);
    unsigned int niter = atoi(argv[3]);

    // Discard the first time running. It apparantly does some extra work during first time
    // JIT??
    run_test_cufft_2d(nx, ny);

    float sum = 0.0;
    for (unsigned int i = 0; i < niter; ++i) {
        sum += run_test_cufft_2d(nx, ny);
    }
    printf("%.6f\n", sum/(float)niter);

    CHECK_CUDA(hipDeviceReset());
    return 0;
}